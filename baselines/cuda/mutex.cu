#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <sys/time.h>

double get_time() {
  struct timeval tv;
  gettimeofday(&tv, nullptr);
  return tv.tv_sec + 1e-6 * tv.tv_usec;
}

constexpr int m = 2;
constexpr int block_size = 128;

struct Node {
  int lock;
  int sum;

  __device__ void inc() {
    /*
    while (atomicCAS(&lock, 0, 1))
      ;
    sum += 1;
    atomicExch(&lock, 0);
     */

    for (int i = 0; i < 32; i++) {
      if (i == threadIdx.x % 32) {
        while (atomicExch(&lock, 1) == 1)
          ;
        // printf("locked\n");
        atomicExch(&lock, 0);
      }
    }
  }
};

__global__ void inc(Node *nodes) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

  int b = i % m;
  nodes[b].inc();

  /*
  int warp_id = threadIdx.x % 32;
  int b = warp_id;
  int done = 0;
  if (true) {
    auto mask = __activemask();
    // printf("mask %d\n", mask);
    while (!__all_sync(mask, done)) {
      for (int k = 0; k < 32; k++) {
        if (k == warp_id && !done) {
          int &lock = nodes[b].lock;
          if (atomicCAS(&lock, 0, 1) == 0) {
            nodes[b].sum += 1;
            done = true;
            atomicExch(&lock, 0);
          }
        }
      }
    }
  } else {
    for (int k = 0; k < 32; k++) {
      if (k == warp_id) {
        int &lock = nodes[b].lock;
        while (atomicCAS(&lock, 0, 1))
          ;
        nodes[b].sum += 1;
        done = true;
        atomicExch(&lock, 0);
      }
    }
  }
  */
}

void mutex() {
  Node *a;

  hipMallocManaged(&a, m * sizeof(Node));

  for (int i = 0; i < 20; i++) {
    hipDeviceSynchronize();
    auto t = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipDeviceSynchronize();
    inc<<<1, 4>>>((Node *)a);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "device  " << milliseconds << std::endl;
    int sum = 0;
    for (int j = 0; j < m; j++) {
      sum += a[j].sum;
    }
    printf("sum %d\n", sum);
  }
  std::cout << std::endl;
}

__global__ void elect(long long *addr_) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  auto addr = addr_[i];
  auto warpId = threadIdx.x % warpSize;

#define FULLMASK 0xFFFFFFFF

  bool has_following_eqiv = 0;
  for (int i = 1; i < warpSize; i++) {
    auto cond = warpId + i < warpSize;
    // auto mask = __ballot_sync(FULLMASK, cond);
    bool same = (addr == __shfl_down_sync(FULLMASK, addr, i));
    if (cond) {
      has_following_eqiv = has_following_eqiv || same;
    }
  }
  if (!has_following_eqiv) {
    printf("%lld\n", addr);
  }
}

void elect_diff() {
  long long *a;

  hipMallocManaged(&a, 32 * sizeof(long long));

  for (int i = 0; i < 32; i++) {
    a[i] = i % 5;
  }

  for (int i = 0; i < 20; i++) {
    hipDeviceSynchronize();
    auto t = get_time();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    hipDeviceSynchronize();
    elect<<<1, 32>>>(a);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "device  " << milliseconds << std::endl;
  }
  std::cout << std::endl;
}

int main() {
  elect_diff();
}
